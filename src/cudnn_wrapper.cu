#include "hip/hip_runtime.h"
#include "hipDNN.h"

#include "common_cuda.h"
#include "cudnn_wrapper.h"

#include <cassert>
#include <vector>

using namespace onnx;
using namespace std;

vector<hipEvent_t> cuda_get_events(int num_events) {
  vector<hipEvent_t> events(0);
  for (int i = 0; i < num_events; ++i) {
    hipEvent_t e;
    CHECK_CUDA(hipEventCreate(&e));
    events.push_back(e);
  }
  return events;
}

PoolingOperator::PoolingOperator(hipdnnHandle_t *handle_,
                                 ValueInfoProto input_shape_, NodeProto node,
                                 hipdnnPoolingMode_t mode)
    : handle{handle_} {

  CHECK_CUDNN(hipdnnCreatePoolingDescriptor(&pool_desc));
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));

  // events = cuda_get_events(2);

  vector<int> shapes(0);
  for (auto d : input_shape_.type().tensor_type().shape().dim()) {
    shapes.push_back(d.dim_value());
  }

  int kernel_h = 0, kernel_w = 0, v_pad = 0, h_pad = 0, v_stride = 0,
      h_stride = 0;
  for (auto attri : node.attribute()) {
    if (attri.name() == "kernel_shape") {
      kernel_h = attri.ints().Get(0);
      kernel_w = attri.ints().Get(1);
    }

    if (attri.name() == "pads") {
      v_pad = attri.ints().Get(2);
      h_pad = attri.ints().Get(3);
    }

    if (attri.name() == "strides") {
      v_stride = attri.ints().Get(0);
      h_stride = attri.ints().Get(1);
    }
  }

  if (kernel_h == 0 || kernel_w == 0) {
    kernel_h = shapes[2];
    kernel_w = shapes[3];
    v_pad = 0;
    h_pad = 0;
    v_stride = 1;
    h_stride = 1;
  }

  CHECK_CUDNN(hipdnnSetPooling2dDescriptor(
      /* hipdnnPoolingDescriptor_t poolingDesc */ pool_desc,
      /* hipdnnPoolingMode_t mode */ mode,
      /* hipdnnNanPropagation_t maxpoolingNanOpt */ HIPDNN_PROPAGATE_NAN,
      /* int windowHeight */ kernel_h,
      /* int windowWidth */ kernel_w,
      /* int verticalPadding */ v_pad,
      /* int horizontalPadding */ h_pad,
      /* int verticalStride */ v_stride,
      /* int horizontalStride */ h_stride));

  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
      /* hipdnnTensorDescriptor_t tensorDesc */ input_desc,
      /* hipdnnTensorFormat_t format */ HIPDNN_TENSOR_NCHW,
      /* hipdnnDataType_t dataType */ HIPDNN_DATA_FLOAT,
      /* int n */ shapes[0],
      /* int c */ shapes[1],
      /* int h */ shapes[2],
      /* int w */ shapes[3]));

  int outN = 0, outC = 0, outH = 0, outW = 0;
  CHECK_CUDNN(hipdnnGetPooling2dForwardOutputDim(
      /* const hipdnnPoolingDescriptor_t poolingDesc */ pool_desc,
      /* const hipdnnTensorDescriptor_t inputDesc */ input_desc,
      /* int *outN */ &outN,
      /* int *outC */ &outC,
      /* int *outH */ &outH,
      /* int *outW */ &outW));

  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
      /* hipdnnTensorDescriptor_t tensorDesc */ output_desc,
      /* hipdnnTensorFormat_t format */ HIPDNN_TENSOR_NCHW,
      /* hipdnnDataType_t dataType */ HIPDNN_DATA_FLOAT,
      /* int n */ outN,
      /* int c */ outC,
      /* int h */ outH,
      /* int w */ outW));
}

void PoolingOperator::dispatch(hipStream_t s) {
  assert(input_is_set && output_is_set);

  float *scalers = new float[2];
  scalers[0] = 1;
  scalers[1] = 0;

  // CHECK_CUDA(hipEventRecord(events[0], s));
  hipdnnSetStream(*handle, s);
  CHECK_CUDNN(hipdnnPoolingForward(
      /* hipdnnHandle_t * handle */ *handle,
      /* const hipdnnPoolingDescriptor_t poolingDesc */ pool_desc,
      /* const void *alpha */ scalers,
      /* const hipdnnTensorDescriptor_t xDesc */ input_desc,
      /* const void *x */ (const void *)(uintptr_t)input,
      /* const void *beta */ scalers + 1,
      /* const hipdnnTensorDescriptor_t yDesc */ output_desc,
      /* void *y */ (void *)(uintptr_t)output));
  // CHECK_CUDA(hipEventRecord(events[1], s));
  // return events;
}

void PoolingOperator::set_argument(KERNEL_ARG arg, hipDeviceptr_t ptr) {

  switch (arg) {
  case (INPUT):
    input = ptr;
    input_is_set = true;
    break;
  case (OUTPUT):
    output = ptr;
    output_is_set = true;
    break;
  default:;
  }
}

AddOperator::AddOperator(hipdnnHandle_t *handle_, ValueInfoProto input_shape_,
                         ValueInfoProto output_shape_)
    : handle{handle_} {
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
  // events = cuda_get_events(2);

  vector<int> input_shapes(0);
  for (auto d : input_shape_.type().tensor_type().shape().dim()) {
    input_shapes.push_back(d.dim_value());
  }

  vector<int> output_shapes(0);
  for (auto d : output_shape_.type().tensor_type().shape().dim()) {
    output_shapes.push_back(d.dim_value());
  }

  for (int i = 0; i < 4; ++i) {
    assert(input_shapes[i] == output_shapes[i]);
    total_size += input_shapes[i];
  }

  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
      /* hipdnnTensorDescriptor_t tensorDesc */ input_desc,
      /* hipdnnTensorFormat_t format */ HIPDNN_TENSOR_NCHW,
      /* hipdnnDataType_t dataType */ HIPDNN_DATA_FLOAT,
      /* int n */ input_shapes[0],
      /* int c */ input_shapes[1],
      /* int h */ input_shapes[2],
      /* int w */ input_shapes[3]));

  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
      /* hipdnnTensorDescriptor_t tensorDesc */ output_desc,
      /* hipdnnTensorFormat_t format */ HIPDNN_TENSOR_NCHW,
      /* hipdnnDataType_t dataType */ HIPDNN_DATA_FLOAT,
      /* int n */ output_shapes[0],
      /* int c */ output_shapes[1],
      /* int h */ output_shapes[2],
      /* int w */ output_shapes[3]));
}

void AddOperator::set_argument(KERNEL_ARG arg, hipDeviceptr_t ptr) {
  if ((arg == INPUT) && (input_is_set)) {
    arg = DATA;
  }
  switch (arg) {
  case (INPUT):
    input = ptr;
    input_is_set = true;
    break;
  case (DATA):
    data = ptr;
    data_is_set = true;
    break;
  case (OUTPUT):
    output = ptr;
    output_is_set = true;
    break;
  default:;
  }
}

void AddOperator::dispatch(hipStream_t s) {
  assert(input_is_set && output_is_set && data_is_set);

  float *scalers = new float[2];
  scalers[0] = 1;
  scalers[1] = 1;

  // CHECK_CUDA(hipEventRecord(events[0], s));
  hipdnnSetStream(*handle, s);

  hipMemcpyDtoDAsync(output, data, sizeof(float) * total_size, s);
  hipdnnAddTensor(
      /* hipdnnHandle_t handle */ *handle,
      /* *alpha */ scalers,
      /* aDesc */ input_desc,
      /* *A */ CUDevicePtrConstCast(input),
      /* *beta */ scalers + 1,
      /* cDesc */ output_desc,
      /* *C */ CUDevicePtrCast(output));

  // CHECK_CUDA(hipEventRecord(events[1], s));
  // return events;
}

ReluOperator::ReluOperator(hipdnnHandle_t *handle_, ValueInfoProto input_shape_)
    : handle{handle_} {
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
  // events = cuda_get_events(2);

  vector<int> input_shapes(0);
  for (auto d : input_shape_.type().tensor_type().shape().dim()) {
    input_shapes.push_back(d.dim_value());
  }

  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
      /* hipdnnTensorDescriptor_t tensorDesc */ input_desc,
      /* hipdnnTensorFormat_t format */ HIPDNN_TENSOR_NCHW,
      /* hipdnnDataType_t dataType */ HIPDNN_DATA_FLOAT,
      /* int n */ input_shapes[0],
      /* int c */ input_shapes[1],
      /* int h */ input_shapes[2],
      /* int w */ input_shapes[3]));

  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
      /* hipdnnTensorDescriptor_t tensorDesc */ output_desc,
      /* hipdnnTensorFormat_t format */ HIPDNN_TENSOR_NCHW,
      /* hipdnnDataType_t dataType */ HIPDNN_DATA_FLOAT,
      /* int n */ input_shapes[0],
      /* int c */ input_shapes[1],
      /* int h */ input_shapes[2],
      /* int w */ input_shapes[3]));

  CHECK_CUDNN(hipdnnCreateActivationDescriptor(&activation_desc));

  CHECK_CUDNN(hipdnnSetActivationDescriptor(
      /* hipdnnActivationDescriptor_t         activationDesc */ activation_desc,
      /* hipdnnActivationMode_t               mode */ HIPDNN_ACTIVATION_RELU,
      /* hipdnnNanPropagation_t               reluNanOpt */ HIPDNN_PROPAGATE_NAN,
      /* double                              coef */ 0.0));
}

void ReluOperator::set_argument(KERNEL_ARG arg, hipDeviceptr_t ptr) {
  switch (arg) {
  case (INPUT):
    input = ptr;
    input_is_set = true;
    break;
  case (OUTPUT):
    output = ptr;
    output_is_set = true;
    break;
  default:;
  }
}

void ReluOperator::dispatch(hipStream_t s) {
  assert(input_is_set && output_is_set);

  float *scalers = new float[2];
  scalers[0] = 1;
  scalers[1] = 1;

  // CHECK_CUDA(hipEventRecord(events[0], s));
  CHECK_CUDNN(hipdnnSetStream(*handle, s));

  CHECK_CUDNN(hipdnnActivationForward(
      /* hipdnnHandle_t handle */ *handle,
      /* hipdnnActivationDescriptor_t     activationDesc */ activation_desc,
      /* const void                     *alpha */ scalers,
      /* const hipdnnTensorDescriptor_t   xDesc */ input_desc,
      /* const void                     *x */ CUDevicePtrConstCast(input),
      /* const void                     *beta */ scalers + 1,
      /* const hipdnnTensorDescriptor_t   yDesc */ output_desc,
      /* void                           *y */ CUDevicePtrCast(output)));

  // CHECK_CUDA(hipEventRecord(events[1], s));
  // return events;
}

BatchNormOperator::BatchNormOperator(hipdnnHandle_t *handle_,
                                     ValueInfoProto input_shape_,
                                     double epsilon_)
    : handle(handle_), epsilon(epsilon_) {
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&batch_norm_desc));
  // events = cuda_get_events(2);

  vector<int> input_shapes(0);
  for (auto d : input_shape_.type().tensor_type().shape().dim()) {
    input_shapes.push_back(d.dim_value());
  }

  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
      /* hipdnnTensorDescriptor_t tensorDesc */ input_desc,
      /* hipdnnTensorFormat_t format */ HIPDNN_TENSOR_NCHW,
      /* hipdnnDataType_t dataType */ HIPDNN_DATA_FLOAT,
      /* int n */ input_shapes[0],
      /* int c */ input_shapes[1],
      /* int h */ input_shapes[2],
      /* int w */ input_shapes[3]));

  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
      /* hipdnnTensorDescriptor_t tensorDesc */ output_desc,
      /* hipdnnTensorFormat_t format */ HIPDNN_TENSOR_NCHW,
      /* hipdnnDataType_t dataType */ HIPDNN_DATA_FLOAT,
      /* int n */ input_shapes[0],
      /* int c */ input_shapes[1],
      /* int h */ input_shapes[2],
      /* int w */ input_shapes[3]));

  CHECK_CUDNN(hipdnnDeriveBNTensorDescriptor(batch_norm_desc, input_desc,
                                            HIPDNN_BATCHNORM_SPATIAL));
}

void BatchNormOperator::set_argument(KERNEL_ARG arg, hipDeviceptr_t ptr) {
  switch (arg) {
  case (INPUT):
    input = ptr;
    input_is_set = true;
    break;
  case (DATA):
    args.push_back(ptr);
    if (args.size() == 4) {
      data_is_set = true;
    }
    break;
  case (OUTPUT):
    output = ptr;
    output_is_set = true;
    break;
  default:;
  }
}

void BatchNormOperator::dispatch(hipStream_t s) {
  assert(input_is_set && data_is_set && output_is_set);

  float *scalers = new float[2];
  scalers[0] = 1;
  scalers[1] = 0;

  // CHECK_CUDA(hipEventRecord(events[0], s));
  CHECK_CUDNN(hipdnnSetStream(*handle, s));

  if (epsilon < 1e-5) {
    epsilon += 1e-6; // handle the edge case where the value is exactly 1e-5
  }

  CHECK_CUDNN(hipdnnBatchNormalizationForwardInference(
      /* hipdnnHandle_t handle */ *handle,
      /* hipdnnBatchNormMode_t mode */ HIPDNN_BATCHNORM_SPATIAL,
      /* const void *alpha */ scalers,
      /* const void *beta */ scalers + 1,
      /* const hipdnnTensorDescriptor_t xDesc */ input_desc,
      /* const void *x */ CUDevicePtrConstCast(input),
      /* const hipdnnTensorDescriptor_t yDesc */ output_desc,
      /* void *y */ CUDevicePtrCast(output),
      /* const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc */
      batch_norm_desc,
      /* const void *bnScale */ CUDevicePtrConstCast(args[0]),
      /* const void *bnBias */ CUDevicePtrConstCast(args[1]),
      /* const void *estimatedMean */ CUDevicePtrConstCast(args[2]),
      /* const void *estimatedVariance */ CUDevicePtrConstCast(args[3]),
      /* double epsilon */ epsilon));

  // CHECK_CUDA(hipEventRecord(events[1], s));
  // return events;
}

SoftMaxOperator::SoftMaxOperator(hipdnnHandle_t *handle_,
                                 ValueInfoProto input_shape_)
    : handle(handle_) {
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
  // events = cuda_get_events(2);

  vector<int> input_shapes(0);
  for (auto d : input_shape_.type().tensor_type().shape().dim()) {
    input_shapes.push_back(d.dim_value());
  }

  if (input_shapes.size() == 2) {
    input_shapes.push_back(1);
    input_shapes.push_back(1);
  }

  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
      /* hipdnnTensorDescriptor_t tensorDesc */ input_desc,
      /* hipdnnTensorFormat_t format */ HIPDNN_TENSOR_NCHW,
      /* hipdnnDataType_t dataType */ HIPDNN_DATA_FLOAT,
      /* int n */ input_shapes[0],
      /* int c */ input_shapes[1],
      /* int h */ input_shapes[2],
      /* int w */ input_shapes[3]));

  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
      /* hipdnnTensorDescriptor_t tensorDesc */ output_desc,
      /* hipdnnTensorFormat_t format */ HIPDNN_TENSOR_NCHW,
      /* hipdnnDataType_t dataType */ HIPDNN_DATA_FLOAT,
      /* int n */ input_shapes[0],
      /* int c */ input_shapes[1],
      /* int h */ input_shapes[2],
      /* int w */ input_shapes[3]));
}

void SoftMaxOperator::set_argument(KERNEL_ARG arg, hipDeviceptr_t ptr) {
  switch (arg) {
  case (INPUT):
    input = ptr;
    input_is_set = true;
    break;
  case (OUTPUT):
    output = ptr;
    output_is_set = true;
    break;
  default:;
  }
}

void SoftMaxOperator::dispatch(hipStream_t s) {
  assert(input_is_set && output_is_set);

  float *scalers = new float[2];
  scalers[0] = 1;
  scalers[1] = 0;

  // CHECK_CUDA(hipEventRecord(events[0], s));
  CHECK_CUDNN(hipdnnSetStream(*handle, s));

  CHECK_CUDNN(hipdnnSoftmaxForward(
      /* hipdnnHandle_t                    handle */ *handle,
      /* hipdnnSoftmaxAlgorithm_t          algorithm */ algo,
      /* hipdnnSoftmaxMode_t               mode */ mode,
      /* const void                      *alpha*/ scalers,
      /* const hipdnnTensorDescriptor_t    xDesc*/ input_desc,
      /* const void                      *x*/ CUDevicePtrConstCast(input),
      /* const void                      *beta*/ scalers + 1,
      /* const hipdnnTensorDescriptor_t    yDesc*/ output_desc,
      /* void                            *y*/ CUDevicePtrCast(output)));

  // CHECK_CUDA(hipEventRecord(events[1], s));
  // return events;
}

ConvOperator::ConvOperator(
    hipdnnHandle_t *handle_, NodeProto node,
    shared_ptr<unordered_map<string, ValueInfoProto>> io_shapes)
    : handle{handle_} {
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  CHECK_CUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));

  // events = cuda_get_events(2);

  string input_name = node.input().Get(0);
  string filter_name = node.input().Get(1);
  string output_name = node.output().Get(0);

  auto shape_vectors = [](ValueInfoProto p) {
    vector<int> shapes(0);
    for (auto d : p.type().tensor_type().shape().dim()) {
      shapes.push_back(d.dim_value());
    }
    return shapes;
  };

  auto input_shapes = shape_vectors(io_shapes->at(input_name));
  auto kernel_shapes = shape_vectors(io_shapes->at(filter_name));
  auto output_shapes = shape_vectors(io_shapes->at(output_name));

  auto attribute_vectors = [](NodeProto n, string attribute, int default_ = 0) {
    vector<int> values;
    for (auto attri : n.attribute()) {
      if (attri.name() == attribute) {
        for (auto val : attri.ints()) {
          values.push_back(val);
        }
      }
    }

    if (values.size() == 0) {
      values.push_back(default_);
      values.push_back(default_);
    }
    return values;
  };

  auto kernel_shape = attribute_vectors(node, "kernel_shape", 0);
  auto pads = attribute_vectors(node, "pads", 0);
  auto strides = attribute_vectors(node, "strides", 1);
  auto dilations = attribute_vectors(node, "dilations", 1);

  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                         /*format=*/HIPDNN_TENSOR_NCHW,
                                         /*dataType=*/HIPDNN_DATA_FLOAT,
                                         /*batch_size=*/input_shapes[0],
                                         /*channels=*/input_shapes[1],
                                         /*image_height=*/input_shapes[2],
                                         /*image_width=*/input_shapes[3]));

  CHECK_CUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                         /*dataType=*/HIPDNN_DATA_FLOAT,
                                         /*format=*/HIPDNN_TENSOR_NCHW,
                                         /*out_channels=*/kernel_shapes[0],
                                         /*in_channels=*/kernel_shapes[1],
                                         /*kernel_height=*/kernel_shapes[2],
                                         /*kernel_width=*/kernel_shapes[3]));

  CHECK_CUDNN(
      hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                      /*pad_height=*/pads[0],
                                      /*pad_width=*/pads[1],
                                      /*vertical_stride=*/strides[0],
                                      /*horizontal_stride=*/strides[1],
                                      /*dilation_height=*/dilations[0],
                                      /*dilation_width=*/dilations[1],
                                      /*mode=*/HIPDNN_CROSS_CORRELATION,
                                      /*computeType=*/HIPDNN_DATA_FLOAT));

  int batch_size{0}, channels{0}, height{0}, width{0};
  CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(
      convolution_descriptor, input_descriptor, kernel_descriptor, &batch_size,
      &channels, &height, &width));
  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                         /*format=*/HIPDNN_TENSOR_NCHW,
                                         /*dataType=*/HIPDNN_DATA_FLOAT,
                                         /*batch_size=*/batch_size,
                                         /*channels=*/channels,
                                         /*image_height=*/height,
                                         /*image_width=*/width));

  assert(batch_size == output_shapes[0]);
  assert(channels == output_shapes[1]);
  assert(height == output_shapes[2]);
  assert(width == output_shapes[3]);
}

void ConvOperator::set_argument(KERNEL_ARG arg, hipDeviceptr_t ptr) {
  switch (arg) {
  case (INPUT):
    input = ptr;
    input_is_set = true;
    break;
  case (DATA):
    if (data_is_set) {
      throw runtime_error("ConvOperator data set twice, maybe a bias term?");
    }
    data = ptr;
    data_is_set = true;
    break;
  case (OUTPUT):
    output = ptr;
    output_is_set = true;
    break;
  default:;
  }
}

void ConvOperator::dispatch(hipStream_t s) {
  assert(input_is_set && output_is_set && data_is_set);

  float *scalers = new float[2];
  scalers[0] = 1;
  scalers[1] = 1;

  // CHECK_CUDA(hipEventRecord(events[0], s));
  CHECK_CUDNN(hipdnnSetStream(*handle, s));
  hipdnnConvolutionForward(
      *handle, scalers, input_descriptor, CUDevicePtrConstCast(input),
      kernel_descriptor, CUDevicePtrConstCast(data), convolution_descriptor,
      HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, scalers + 1,
      output_descriptor, CUDevicePtrCast(output));

  // CHECK_CUDA(hipEventRecord(events[1], s));
  // return events;
}