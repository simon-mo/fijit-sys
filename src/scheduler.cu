//
// Created by Simon Mo on 2019-03-21.
//

#include "common_cuda.h"
#include "scheduler.h"

#include <chrono>
#include <iostream>
#include <memory>
#include <thread>

using namespace std;

shared_ptr<BlockingConcurrentQueue<shared_ptr<PhysicalOperator>>>
Scheduler::register_model_queue(
        string model_name,
shared_ptr<BlockingConcurrentQueue<shared_ptr<LogicalOperator>>> q) {
logical_op_queues.insert({model_name, q});

shared_ptr<BlockingConcurrentQueue<shared_ptr<PhysicalOperator>>> ops_q =
make_shared<BlockingConcurrentQueue<shared_ptr<PhysicalOperator>>>();

physical_op_queues.insert({model_name, ops_q});

return ops_q;
}

void Scheduler::register_total_resource(shared_ptr<int> total_resource_estimate) {
    total_resource = total_resource_estimate;
}

void Scheduler::stop() { shouldStop = true; }

void StaticScheduler::start() {
    CHECK_CUDEVICE(hipCtxSetCurrent(*ctx));

    while (true) {
        if (shouldStop) {
            break;
        }
        schedule();
    }
}

StaticScheduler::StaticScheduler(int max_blocks_per_model, hipCtx_t *ctx,
                                 hipdnnHandle_t *handle_, hipblasHandle_t *cublasHandle_)
        : max_blocks(max_blocks_per_model), ctx(ctx), handle(handle_), cublasHandle(cublasHandle_) {}

void StaticScheduler::schedule() {
    auto num_models = logical_op_queues.size();

  auto num_models = logical_op_queues.size();

  if (num_models * max_blocks > *total_resource) {
    cerr << "StaticScheduler::schedule allocated resource exceeds current "
            "total resource, skipping..."
         << endl;
    return;
  }

  for (auto &entry : logical_op_queues) {
    string model_name = entry.first;
    shared_ptr<ConcurrentQueue<shared_ptr<PhysicalOperator>>> dispatch_queue =
        physical_op_queues.at(model_name);

    shared_ptr<ConcurrentQueue<shared_ptr<LogicalOperator>>> op_queue =
        entry.second;

    shared_ptr<LogicalOperator> op;
    while (op_queue->try_dequeue(op)) {
      shared_ptr<PhysicalOperator> physical_op =
          op->realize(max_blocks, handle, cublasHandle);
      bool success = dispatch_queue->enqueue(physical_op);

    int num_entries = 0;
    for (auto &entry : logical_op_queues) {
        string model_name = entry.first;
        shared_ptr<BlockingConcurrentQueue<shared_ptr<PhysicalOperator>>> dispatch_queue =
                physical_op_queues.at(model_name);

        shared_ptr<BlockingConcurrentQueue<shared_ptr<LogicalOperator>>> op_queue =
                entry.second;

        // cerr << model_name << " queue size " << op_queue->size_approx() << endl;

        while (true) {
            shared_ptr<LogicalOperator> ops[1024];
            const int num_dequeued = op_queue->wait_dequeue_bulk_timed(ops, 1024, std::chrono::milliseconds(100));
            if (num_dequeued == 0)
                break;

            for (int i = 0; i < num_dequeued; i++) {
                shared_ptr<LogicalOperator> op = ops[i];
                shared_ptr<PhysicalOperator> physical_op = op->realize(max_blocks, handle, cublasHandle);
                bool success = dispatch_queue->enqueue(physical_op);

                if (!success) {
                    cerr << "Failed to enqueue operation to dispatch queue" << endl;
                }
                num_entries++;
            }
        }
    }

    // cerr << "\t[StaticScheduler] Loaded " << num_entries << " physical operations" << endl;
}
