#include "common_cuda.h"
#include "hip/hip_runtime.h"
#include "executor.h"

#include <chrono>
#include <iostream>
#include <thread>
#include <tuple>

#include "fmt/core.h"
#include "glog/logging.h"

using namespace std;

void Executor::register_queue(string model_name, PhysicalOpQueue queue) {
  hipStream_t s;
  hipStreamCreate(&s);
  ExecutorCtx ctx = {model_name, s, queue};
  executor_queues.emplace_back(ctx);
}

void Executor::stop() {
  wait();
  should_stop = true;
}

void Executor::start() {
  CHECK_CUDEVICE(hipCtxSetCurrent(*ctx));

  while (true) {
    if (should_stop) {
      break;
    }

    shared_ptr<PhysicalOperator> op = nullptr;

    int tid_counter = 0;
    for (ExecutorCtx &ctx_struct : executor_queues) {
      tid_counter++;

      // while (ctx_struct.queue->try_dequeue(op)) {
      if (!ctx_struct.queue->try_dequeue(op)) {
        continue;
      }
      string op_name =
          fmt::format("{}-{}", ctx_struct.model_name, op->get_name());
      // events_registrar.record(EventType::BEGIN, EventSource::Executor,
      //                         op_name);
      if (op->is_timing && op->event_type == EventType::BEGIN) {
        events_registrar.record(EventType::BEGIN, EventSource::GPU, op_name,
                                tid_counter, ctx_struct.stream);
      }

      op->dispatch(ctx_struct.stream);

      if (op->is_timing && op->event_type == EventType::END) {
        events_registrar.record(EventType::END, EventSource::GPU, op_name,
                                tid_counter, ctx_struct.stream);
      }

      // events_registrar.record(EventType::END, EventSource::Executor,
      // op_name);
      // }
    }
  }
}

void Executor::wait() {
  for (ExecutorCtx &ctx_struct : executor_queues) {
    while (ctx_struct.queue->size_approx() != 0) {
      std::this_thread::sleep_for(10ms);
    }
  }
  hipDeviceSynchronize();
}