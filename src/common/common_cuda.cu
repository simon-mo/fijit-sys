//
// Created by Simon Mo on 2019-03-13.
//
#include "common/common_cuda.h"
#include <chrono>
#include <vector>

using namespace std;

hipCtx_t cuda_init() {
  hipDevice_t cuDevice;
  hipCtx_t cuContext;
  CHECK_CUDEVICE(hipInit(0));
  CHECK_CUDEVICE(hipDeviceGet(&cuDevice, 0));
  CHECK_CUDEVICE(hipCtxCreate(&cuContext, 0, cuDevice));
  return cuContext;
}

void CUDART_CB host_record_time(hipStream_t stream, hipError_t status,
                                void *data) {
  int64_t *buf = (int64_t *)data;
  *buf = std::chrono::high_resolution_clock::now().time_since_epoch().count();
}
